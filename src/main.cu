#include "common.hpp"
#include "jobgenerator.hpp"
#include "sha1_hash_search.cuh"

#include <stdint.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <vector>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

constexpr u32 GRID_SIZE = 1024;
constexpr u32 BLOCK_SIZE = 256;

int main() {
    initialize_cuda_constants();

    int min_grid_size, block_size;

    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, search_block, 0, 0);

    //u32 N = min_grid_size * 2;
    //u32 M = block_size / 2;

    u32 N = 256;
    u32 M = 256;

    printf("grid: %u, block: %u\n", N, M);

    hipStream_t stream[4];
    for (u32 i = 0; i < 4; i++) {
        CHECK_CUDA_ERROR(hipStreamCreate(&stream[i]));
    }

    uint8_t *payload;
    CHECK_CUDA_ERROR(hipMallocManaged(&payload, N * M * 64));

    char const *header = "toteload/davidbos+dot+me/";
    u32 header_len = strlen(header);

    JobGenerator generator(55 - header_len);

    std::vector<JobDescription> jobs;
    jobs.resize(N * M);

    uint32_t *out;
    CHECK_CUDA_ERROR(hipMallocManaged(&out, N * M * 5 * sizeof(u32)));

    u32 *idx;
    CHECK_CUDA_ERROR(hipMallocManaged(&idx, N * M * 3 * sizeof(u32)));

    u32 best_hash[5] = { 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, };

    auto start = std::chrono::high_resolution_clock::now();

    for (u32 i = 0; i < 12; i++) {
        for (u32 i = 0; i < N * M; i++) {
            generator.next(jobs[i]);
        }

        for (u32 i = 0; i < jobs.size(); i++) {
            write_payload(jobs[i], header, header_len, payload + i * 64);
        }

        for (u32 i = 0; i < jobs.size(); i++) {
            idx[i*3+0] = jobs[i].search_idxs[0];
            idx[i*3+1] = jobs[i].search_idxs[1];
            idx[i*3+2] = jobs[i].search_idxs[2];
        }

        search_block<<<N, M>>>(payload, idx, out);

        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        for (int i = 0; i < N * M; i++) {
            u32 *candidate = out + i * 5;
            if (is_better_hash(best_hash, candidate)) {
                memcpy(best_hash, candidate, 20);
            }
        }

        for (int i = 0; i < 5; i++) {
            printf("%08x ", best_hash[i]);
        }

        puts("");
    }

    auto end = std::chrono::high_resolution_clock::now();
    auto d = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

    printf("duration %f s\n", d.count());
    printf("%f MH/s\n", (double)(64*64*64) * 12 * N * M / 1'000'000.0 / d.count());

    //printf("%.55s\n", payload);

    hipFree(payload);
    hipFree(out);

    return 0;
}

